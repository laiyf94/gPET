#include "hip/hip_runtime.h"
#ifndef __FINALIZE_CU__
#define __FINALIZE_CU__

void outputData(const char* srcname, const int size, const char* outputfilename, const char* mode)
//      output data to file
{
    void* tempData;
    hipMalloc((void**)&tempData, size);
    if (hipMemcpyFromSymbol(tempData, HIP_SYMBOL(srcname), size, 0, hipMemcpyDeviceToDevice) != hipSuccess)
        cout << "error in getting " << srcname << endl;

    outputData(tempData, size, outputfilename, mode);
    hipFree(tempData);
}

void outputData(void* src, const int size, const char* outputfilename, const char* mode)
//      output data to file
{
    //      copy data from GPU to CPU
    void* tempData_h = malloc(size);
    hipMemcpy(tempData_h, src, size, hipMemcpyDeviceToHost);
    //  cout << "out put data ... mode " << mode <<" filename "<< outputfilename<< endl;
    //      write results to file
    FILE* fp;
    fp = fopen(outputfilename, mode);
    if (fp == NULL) {
        cout << "Can not open file to write results.";
        exit(1);
    }
    fwrite(tempData_h, size, 1, fp);
    fclose(fp);

    //      free space
    free(tempData_h);
}

void fina()
/*******************************************************************
c*    finalizes the gCTD system
c******************************************************************/
{
    //free memories
    //      mark the start time
    time_t start_time, end_time;
    float time_diff;

    start_time = clock();

    //      printf any results during computing
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    //      output dose results

    //      outputData("fEscore", sizeof(float)*NXYZ, outputAveName, "wb");
    //      outputData("fEscor2", sizeof(float)*NXYZ, outputStdName, "wb");

    //      free lamph info
    hipUnbindTexture(lamph_tex);
    hipFreeArray(lamph);

    //      free cmpt info
    hipUnbindTexture(compt_tex);
    hipFreeArray(compt);
    hipUnbindTexture(s_tex);
    hipFreeArray(sArray);

    //      free phote info
    hipUnbindTexture(phote_tex);
    hipFreeArray(phote);

    //      free rayle info
    hipUnbindTexture(rayle_tex);
    hipFreeArray(rayle);
    hipUnbindTexture(f_tex);
    hipFreeArray(fArray);

    //      free mat, struc, and dens info
    hipUnbindTexture(mat_tex);
    hipFreeArray(mat);

    hipUnbindTexture(dens_tex);
    hipFreeArray(dens);

    //      free texture file for electron transport
    hipUnbindTexture(scssp_tex);
    hipFreeArray(scssp);
    //      free st info
    hipUnbindTexture(stsp_tex);
    hipFreeArray(stsp);
    //      free scp info
    hipUnbindTexture(scpsp_tex);
    hipFreeArray(scpsp);
    //      free qsurf info
    hipUnbindTexture(q_tex);
    hipFreeArray(qArray);
    //      free bwsp info
    hipUnbindTexture(bwsp_tex);
    hipFreeArray(bwsp);

    //      free wck info
    hipUnbindTexture(woock_tex);
    hipFreeArray(woock);

    hipUnbindTexture(icdf_ZDist_tex);
    hipFreeArray(icdf_ZDist);
    hipUnbindTexture(icdf_EDist_tex);
    hipFreeArray(icdf_EDist);
    hipUnbindTexture(icdf_PhiDist_tex);
    hipFreeArray(icdf_PhiDist);

    hipUnbindTexture(mear_tex);
    hipFreeArray(mear);

    hipFree(escore);
    /*  hipFree(fEscore);
        hipFree(fEscor2); */

    hipFree(x_phap_gBrachy);
    hipFree(vx_phap_gBrachy);

    //      mark the end timer
    printf("\n");
    printf("\n");
    printf("Finalize: Done.\n");

    end_time = clock();
    time_diff = ((float)end_time - (float)start_time) / 1000.0;
    printf("\n\n****************************************\n");
    printf("Finalization time: %f ms.\n\n", time_diff);
    printf("****************************************\n\n\n");
}

PatientDose getDose()
{

    PatientDose patientDose;
    patientDose.doseAve = new float[NXYZ];
    // patientDose.doseStd =  new float[NXYZ];

    //        void *tempData;
    //        hipMalloc( (void **) &tempData, NXYZ*sizeof(float));
    //
    //        if( hipMemcpyFromSymbol(tempData, HIP_SYMBOL("fEscore"), NXYZ*sizeof(float), 0, hipMemcpyDeviceToDevice) != hipSuccess)
    //        cout << "error in getting fEscore" << endl;
    //        hipMemcpy(patientDose.doseAve, tempData, NXYZ*sizeof(float), hipMemcpyDeviceToHost) ;
    //
    //        if( hipMemcpyFromSymbol(tempData, HIP_SYMBOL("fEscor2"), NXYZ*sizeof(float), 0, hipMemcpyDeviceToDevice) != hipSuccess)
    //        cout << "error in getting fEscore2" << endl;
    //        hipMemcpy(patientDose.doseStd, tempData, NXYZ*sizeof(float), hipMemcpyDeviceToHost) ;

    //hipMemcpy(patientDose.doseAve, fEscore, NXYZ*sizeof(float), hipMemcpyDeviceToHost) ;
    // hipMemcpy(patientDose.doseStd, fEscor2, NXYZ*sizeof(float), hipMemcpyDeviceToHost) ;

    hipMemcpy(patientDose.doseAve, escore, NXYZ * sizeof(float), hipMemcpyDeviceToHost);

    patientDose.totalParticleWeight = totalWeight_gBrachy;

    //        hipFree(tempData);

    return patientDose;
}

#endif
